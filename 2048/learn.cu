#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>
#include "board.hpp"
#include "attr.hpp"

#include <vector>

#include <time.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define INF 0x71227122

#define gpuErrchk(ans) { gpuAssert((ans),__FILE__,__LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line)
{
	if (code != hipSuccess){
		fprintf(stderr, "error: %s %s %d\n", hipGetErrorString(code), file, line);
	}

}

using namespace std;

/*
 * all variable with suffix _d means it's a device memory(global memory)
 * all variable with suffix _ds means it's a device share memory
 */


typedef int (board::*MoveFunc)(bool);
// record the two continuous states in the game
class record {
public:
	record() {}
	record(board _s1, board _s2, int _earned) :s1(_s1), s2(_s2), earned(_earned) {}
	board s1, s2;
	int earned;
};

class record_d {
public:
	int s[2][4];
	int earned;
};

int recProgress_h[10005][3];

const int recordSize = 50000;		// should larger than any possible game's length
const int attrNumLimit = 4;			// array size
const int attrNum = 4;				// actual usage
const int attrSlotNum = 4;			// the number of slots in any attribute
const int attrDataSize = attrSlotNum*4;// the table size of attribute
const int branchSizeLimit = 1000;	// the number of random seed
const int branchSize = 208;			// the maximum number of thread (the number of rec array) should be larger than the maximum number of block GPU can run simultaneously

vector<record> rec;

// read in attribute data
bool load(const char* filename, vector<Attr> &attr) {
	FILE* fin = fopen(filename, "rb");
	if (!fin){
		return false;
	}
	int attrN;
	fread(&attrN, sizeof(int), 1, fin);
	attr.resize(attrN);
	for (int i = 0; i<attrN; i++){
		fread(&attr[i].slotNum, sizeof(int), 1, fin);
		fread(&attr[i].position, sizeof(int), 1, fin);
		attr[i].data = new float[1 << (attr[i].slotNum << 2)];
		fread(&(*attr[i].data), sizeof(float), 1 << (attr[i].slotNum << 2), fin);
	}
	fclose(fin);
	return true;
}
// write out attribute data
bool save(const char* filename, vector<Attr> &attr) {
	FILE* fout = fopen(filename, "wb");
	if (!fout){
		return false;
	}
	int attrN = attr.size();
	fwrite(&attrN, sizeof(int), 1, fout);
	for (int i = 0; i<attrN; i++){
		fwrite(&attr[i].slotNum, sizeof(int), 1, fout);
		fwrite(&attr[i].position, sizeof(int), 1, fout);
		fwrite(&(*attr[i].data), sizeof(float), 1 << (attr[i].slotNum << 2), fout);
		delete[] attr[i].data;
	}
	fclose(fout);
	attr.clear();
	return true;
}

void deepCopy(vector<Attr> &dst, vector<Attr> &src) {
	dst.clear();
	dst.resize(src.size());
	for (int i = 0; i<(int)src.size(); i++){
		dst[i].slotNum = src[i].slotNum;
		dst[i].position = src[i].position;
		dst[i].data = new float[1 << (src[i].slotNum << 2)];
		memcpy(dst[i].data, src[i].data, sizeof(float)*(1 << (src[i].slotNum << 2)));
	}
}

void attrDestroy(vector<Attr> &attr) {
	for (int i = 0; i<(int)attr.size(); i++){
		delete[] attr[i].data;
	}
	attr.clear();
}

// initialize random seed
__forceinline__ __global__ void initrand(hiprandState_t *randState, int *randSeed_d)
{
	int blockID = blockIdx.x;
	hiprand_init(randSeed_d[blockID], 0, 0, &randState[blockID]);
}

/*
 * high      low bit
 *  0  1  2  3
 *  4  5  6  7
 *  8  9  10 11
 *  12 13 14 15
 */

// get a single cell in a board
__forceinline__ __device__ int getCell_d(int *board_ds, int x, int y)
{
	return (board_ds[x] >> ((3 - y) << 2)) & 0xf;
}
// set a single cell in a board
__forceinline__ __device__ void setCell_d(int *board_ds, int x, int y, int val)
{
	board_ds[x] = (board_ds[x] & (0xffff ^ (0xf << ((3 - y) << 2)))) | (val << ((3 - y) << 2));
}

// generate a random number ( should be called only when there is at least one empty cell )
__forceinline__ __device__ void genCell_d(int *board_ds, hiprandState_t *randState)
{
	unsigned long long pos = 0;
	int cnt = 0;
	for (unsigned long long i = 0; i<16; i++)		// collect all empty positions
		if (!getCell_d(board_ds, (int)(i >> 2), (int)(i & 0x3)))
			pos |= i << (cnt++ << 2);
	if (cnt != 0){
		int num = ((hiprand(randState) & 0x7fffffff) % 10 == 0) ? 2 : 1; //2:4 = 9:1
		int tar = (hiprand(randState) & 0x7fffffff) % cnt;
		tar = (pos >> (tar << 2)) & 0xf;
		setCell_d(board_ds, tar >> 2, tar & 0x3, num);
	}
}

// push a row from right to left, which is higher bit
__forceinline__ __device__ void pushLeft_d(int &n) {
	if ((n & 0xf0) == 0)
		n = (n & 0xff00) | ((n & 0xf) << 4);
	if ((n & 0xf00) == 0)
		n = (n & 0xf000) | ((n & 0xff) << 4);
	if ((n & 0xf000) == 0)
		n <<= 4;
}
// push a board from right to left and return the moving score
__forceinline__ __device__ int left_d(int &board_ds) {
	int ori = board_ds;
	int scoreSum = 0;
	pushLeft_d(board_ds);
	if ((board_ds & 0xf000) && ((board_ds ^ (board_ds << 4)) & 0xf000) == 0){
		scoreSum += (2 << ((board_ds & 0xf000) >> 12));
		board_ds = (board_ds & 0xf0ff) + 0x1000;
	}
	if ((board_ds & 0xf00) && ((board_ds ^ (board_ds << 4)) & 0xf00) == 0){
		scoreSum += (2 << ((board_ds & 0xf00) >> 8));
		board_ds = (board_ds & 0xff0f) + 0x100;
	}
	if ((board_ds & 0xf0) && ((board_ds ^ (board_ds << 4)) & 0xf0) == 0){
		scoreSum += (2 << ((board_ds & 0xf0) >> 4));
		board_ds = (board_ds & 0xfff0) + 0x10;
	}
	pushLeft_d(board_ds);
	if (ori == board_ds)
		return -1;
	else
		return scoreSum;
}

// make a row flip horizontally
__forceinline__ __device__ void mirrorLR_d(int &n)
{
	n = ((n & 0xf) << 12) | ((n & 0xf0) << 4) | ((n & 0xf00) >> 4) | (n >> 12);
}
// make a row flip diagonally. the complicate assignment is to avoid bank comflict, which may not be necessary to that complicate.
__forceinline__ __device__ void diagonal_d(int *board_ds, int attrID)
{
	int tmp = 0;
	tmp |= ((board_ds[(attrID + 0) & 0x3] >> (attrID << 2)) & 0xf) << (((attrID + 0) & 0x3) << 2);
	tmp |= ((board_ds[(attrID + 1) & 0x3] >> (attrID << 2)) & 0xf) << (((attrID + 1) & 0x3) << 2);
	tmp |= ((board_ds[(attrID + 2) & 0x3] >> (attrID << 2)) & 0xf) << (((attrID + 2) & 0x3) << 2);
	tmp |= ((board_ds[(attrID + 3) & 0x3] >> (attrID << 2)) & 0xf) << (((attrID + 3) & 0x3) << 2);
	board_ds[attrID] = tmp;
}
// make a row flip anti-diagonally. the complicate assignment is to avoid bank comflict, which may not be necessary to that complicate.
__forceinline__ __device__ void adiagonal_d(int *board_ds, int attrID)
{
	int tmp = 0;
	tmp |= ((board_ds[(attrID + 0) & 0x3] >> (attrID << 2)) & 0xf) << (((7 - attrID - 0) & 0x3) << 2);
	tmp |= ((board_ds[(attrID + 1) & 0x3] >> (attrID << 2)) & 0xf) << (((7 - attrID - 1) & 0x3) << 2);
	tmp |= ((board_ds[(attrID + 2) & 0x3] >> (attrID << 2)) & 0xf) << (((7 - attrID - 2) & 0x3) << 2);
	tmp |= ((board_ds[(attrID + 3) & 0x3] >> (attrID << 2)) & 0xf) << (((7 - attrID - 3) & 0x3) << 2);
	board_ds[3 - attrID] = tmp;
}

// get the score of a board from attribute
__forceinline__ __device__ void getScore_d(float *attr_d, int *attrPosition_ds, float *getScore_ds, int *board_ds, int attrID)
{
	int id = 0;
	for (int j = 0; j<attrSlotNum; j++){
		int pos = (attrPosition_ds[attrID] >> (j << 2)) & 0xf;
		id |= getCell_d(board_ds, pos >> 2, pos & 0x3) << (j << 2);
	}
	getScore_ds[attrID] = attr_d[attrID*(1 << attrDataSize) + id];
}
// update the score of a board
__forceinline__ __device__ void updateAttr_d(float *attr_d, int *attrPosition_ds, int *board_ds, int attrID, float val)
{
	int id = 0;
	for (int j = 0; j<attrSlotNum; j++){
		int pos = (attrPosition_ds[attrID] >> (j << 2)) & 0xf;
		id |= getCell_d(board_ds, pos >> 2, pos & 0x3) << (j << 2);
	}
	atomicAdd(&attr_d[attrID*(1 << attrDataSize) + id], val);
}

// the main loop of learning
__global__ void run_d(float *attr_d, int *attrPosition_d, record_d *rec_d, hiprandState_t *randState, float learnSpeed, int *learnCnt_d, int *recProgress_d)
{
	bool learn = learnSpeed > 1e-6;
	__shared__ int board_ds[8][4][4];	// 8 way, 4 direction, 4 row
	__shared__ int earnScore_ds[4][4];	// 4 direction, 4 row
	__shared__ float getScore_ds[8][4][attrNumLimit];	// 8 way, 4 direction, 4 attr
	__shared__ int attrPosition_ds[attrNumLimit];	// 4 attr
	__shared__ hiprandState_t randState_ds;
	__shared__ int tar_ds;
	__shared__ int activeID;
	// initialization
	int attrID = threadIdx.x % 4 + (threadIdx.x / 4 / 4 / 8) * 4;
	int directID = threadIdx.x / 4 % 4;
	int wayID = threadIdx.x / 4 / 4 % 8;
	if (attrID < 4) {
		board_ds[wayID][directID][attrID] = 0;
	}
	if (wayID == 0 && directID == 0)
		attrPosition_ds[attrID] = attrPosition_d[attrID];
	__syncthreads();
	int score = 0;
	int step = 1;
	if (threadIdx.x == 0){
		int oldcnt = atomicAdd(&(learnCnt_d[1]), 1);
		activeID = oldcnt%branchSize;
		randState_ds = randState[oldcnt % 1000];
		genCell_d(board_ds[0][0], &randState_ds);
		genCell_d(board_ds[0][0], &randState_ds);
	}
	__syncthreads();
	do{
		// build 4 direction once
		if (wayID == 0 && attrID<4){
			if (directID>0){
				board_ds[0][directID][attrID] = board_ds[0][0][attrID];
			}
			// up; down; left; right;
			if (directID == 0){
				adiagonal_d(board_ds[0][0], attrID);
				diagonal_d(board_ds[0][1], attrID);
				mirrorLR_d(board_ds[0][3][attrID]);
			}
			// push all four boards to left and get their score
			earnScore_ds[directID][attrID] = left_d(board_ds[0][directID][attrID]);
			if (attrID == 0){	// checking if the move is legal
				int unchange = 0;
				int tmpsum = 0;
				for (int i = 0; i<4; i++){
					if (earnScore_ds[directID][i] == -1){
						unchange++;
					}
					else{
						tmpsum += earnScore_ds[directID][i];
					}
				}
				earnScore_ds[directID][0] = unchange == 4 ? -1 : tmpsum;
			}
			if (directID == 0){
				adiagonal_d(board_ds[0][0], attrID);
				diagonal_d(board_ds[0][1], attrID);
				mirrorLR_d(board_ds[0][3][attrID]);
			}
		}
		__syncthreads();
		// copy for 8 way
		if (wayID>0 && attrID<4){
			board_ds[wayID][directID][attrID] = board_ds[0][directID][attrID];
		}
		__syncthreads();
		// flip 8 to different directions
		if (attrID<4){
			if (wayID == 1){
				mirrorLR_d(board_ds[wayID][directID][attrID]);
			}
			else if (wayID == 2){
				board_ds[wayID][directID][attrID] = board_ds[wayID][directID][3 - attrID];
			}
			else if (wayID == 3){
				mirrorLR_d(board_ds[wayID][directID][attrID]);
				board_ds[wayID][directID][attrID] = board_ds[wayID][directID][3 - attrID];
			}
			else if (wayID == 4){
				diagonal_d(board_ds[wayID][directID], attrID);
			}
			else if (wayID == 5){
				diagonal_d(board_ds[wayID][directID], attrID);
				mirrorLR_d(board_ds[wayID][directID][attrID]);
			}
			else if (wayID == 6){
				diagonal_d(board_ds[wayID][directID], attrID);
				board_ds[wayID][directID][attrID] = board_ds[wayID][directID][3 - attrID];
			}
			else if (wayID == 7){
				diagonal_d(board_ds[wayID][directID], attrID);
				mirrorLR_d(board_ds[wayID][directID][attrID]);
				board_ds[wayID][directID][attrID] = board_ds[wayID][directID][3 - attrID];
			}
		}
		__syncthreads();
		// calculate getscore
		if (earnScore_ds[directID][0] != -1){
			getScore_d(attr_d, attrPosition_ds, getScore_ds[wayID][directID], board_ds[wayID][directID], attrID);
			if ((wayID & 0x1) == 0 && (attrID & 0x3) == 0){
				getScore_ds[wayID][directID][attrID] += getScore_ds[wayID][directID][attrID + 1];
				getScore_ds[wayID][directID][attrID] += getScore_ds[wayID][directID][attrID + 2];
				getScore_ds[wayID][directID][attrID] += getScore_ds[wayID][directID][attrID + 3];
				getScore_ds[wayID][directID][attrID] += getScore_ds[wayID + 1][directID][attrID + 0];
				getScore_ds[wayID][directID][attrID] += getScore_ds[wayID + 1][directID][attrID + 1];
				getScore_ds[wayID][directID][attrID] += getScore_ds[wayID + 1][directID][attrID + 2];
				getScore_ds[wayID][directID][attrID] += getScore_ds[wayID + 1][directID][attrID + 3];
			}
		}
		__syncthreads();
		if (earnScore_ds[directID][0] != -1){
			if (wayID == 0 && attrID == 0){
				float tmpScore = 0;
				for (int i = 0; i<8; i += 2){
					for (int j = 0; j<attrNum; j += 4){
						tmpScore += getScore_ds[i][directID][j];
					}
				}
				getScore_ds[0][directID][0] = tmpScore + earnScore_ds[directID][0];
			}
		}
		__syncthreads();
		// choose the best move
		if (wayID == 0 && directID == 0 && attrID == 0){
			tar_ds = -1;
			float tarScore = -100;
			for (int i = 0; i<4; i++){
				if (earnScore_ds[i][0] != -1){
					if (tar_ds == -1 || getScore_ds[0][i][0]>tarScore){
						tar_ds = i;
						tarScore = getScore_ds[0][i][0];
					}
				}
			}
		}
		__syncthreads();
		if (tar_ds == -1) break;		// break out the main loop if die
		if (wayID == 0 && directID == 0 && attrID<4){		// record the game states
			if (learn) {
				rec_d[activeID*recordSize + step - 1].s[1][attrID] = board_ds[0][tar_ds][attrID];
				if (attrID == 0){
					rec_d[activeID*recordSize + step - 1].earned = earnScore_ds[tar_ds][0];
				}
				rec_d[activeID*recordSize + step++].s[0][attrID] = board_ds[0][tar_ds][attrID];
			}
			board_ds[0][0][attrID] = board_ds[0][tar_ds][attrID];
		}
		else{
			step++;
		}
		__syncthreads();
		if (threadIdx.x == 0){
			score += earnScore_ds[tar_ds][0];
			genCell_d(board_ds[0][0], &randState_ds);
		}
		__syncthreads();
	} while (1);
	if (learn) {
		float dif;
		// update the last state
		// copy for 8 way
		if (directID == 0 && attrID<4){
			board_ds[wayID][directID][attrID] = rec_d[activeID*recordSize + step - 2].s[1][attrID];
		}
		__syncthreads();
		if (directID == 0 && attrID<4){
			if (wayID == 1){
				mirrorLR_d(board_ds[wayID][directID][attrID]);
			}
			else if (wayID == 2){
				board_ds[wayID][directID][attrID] = board_ds[wayID][directID][3 - attrID];
			}
			else if (wayID == 3){
				mirrorLR_d(board_ds[wayID][directID][attrID]);
				board_ds[wayID][directID][attrID] = board_ds[wayID][directID][3 - attrID];
			}
			else if (wayID == 4){
				diagonal_d(board_ds[wayID][directID], attrID);
			}
			else if (wayID == 5){
				diagonal_d(board_ds[wayID][directID], attrID);
				mirrorLR_d(board_ds[wayID][directID][attrID]);
			}
			else if (wayID == 6){
				diagonal_d(board_ds[wayID][directID], attrID);
				board_ds[wayID][directID][attrID] = board_ds[wayID][directID][3 - attrID];
			}
			else if (wayID == 7){
				diagonal_d(board_ds[wayID][directID], attrID);
				mirrorLR_d(board_ds[wayID][directID][attrID]);
				board_ds[wayID][directID][attrID] = board_ds[wayID][directID][3 - attrID];
			}
		}
		__syncthreads();
		// calculate getscore
		if (directID == 0){
			getScore_d(attr_d, attrPosition_ds, getScore_ds[wayID][directID], board_ds[wayID][directID], attrID);
			if ((wayID & 0x1) == 0 && (attrID & 0x3) == 0){
				getScore_ds[wayID][directID][attrID] += getScore_ds[wayID][directID][attrID + 1];
				getScore_ds[wayID][directID][attrID] += getScore_ds[wayID][directID][attrID + 2];
				getScore_ds[wayID][directID][attrID] += getScore_ds[wayID][directID][attrID + 3];
				getScore_ds[wayID][directID][attrID] += getScore_ds[wayID + 1][directID][attrID + 0];
				getScore_ds[wayID][directID][attrID] += getScore_ds[wayID + 1][directID][attrID + 1];
				getScore_ds[wayID][directID][attrID] += getScore_ds[wayID + 1][directID][attrID + 2];
				getScore_ds[wayID][directID][attrID] += getScore_ds[wayID + 1][directID][attrID + 3];
			}
		}
		__syncthreads();
		if (directID == 0){
			if (wayID == 0 && attrID == 0){		// could be parallel for wayID
				float tmpScore = 0;
				for (int i = 0; i<8; i += 2){
					for (int j = 0; j<attrNum; j += 4){
						tmpScore += getScore_ds[i][directID][j];
					}
				}
				getScore_ds[0][directID][0] = tmpScore;
			}
		}
		__syncthreads();
		if (directID == 0){
			dif = 0 - getScore_ds[0][0][0];
			updateAttr_d(attr_d, attrPosition_ds, board_ds[wayID][directID], attrID, dif*learnSpeed);
		}
		__syncthreads();
		// update all the state from the end of game to the beginning
		for (int i = step - 2; i>0; i--){
			float s[2];
			// copy for 8 way
			if (directID<2 && attrID<4){
				board_ds[wayID][directID][attrID] = rec_d[activeID*recordSize + i].s[directID][attrID];
			}
			__syncthreads();
			if (directID<2 && attrID<4){
				if (wayID == 1){
					mirrorLR_d(board_ds[wayID][directID][attrID]);
				}
				else if (wayID == 2){
					board_ds[wayID][directID][attrID] = board_ds[wayID][directID][3 - attrID];
				}
				else if (wayID == 3){
					mirrorLR_d(board_ds[wayID][directID][attrID]);
					board_ds[wayID][directID][attrID] = board_ds[wayID][directID][3 - attrID];
				}
				else if (wayID == 4){
					diagonal_d(board_ds[wayID][directID], attrID);
				}
				else if (wayID == 5){
					diagonal_d(board_ds[wayID][directID], attrID);
					mirrorLR_d(board_ds[wayID][directID][attrID]);
				}
				else if (wayID == 6){
					diagonal_d(board_ds[wayID][directID], attrID);
					board_ds[wayID][directID][attrID] = board_ds[wayID][directID][3 - attrID];
				}
				else if (wayID == 7){
					diagonal_d(board_ds[wayID][directID], attrID);
					mirrorLR_d(board_ds[wayID][directID][attrID]);
					board_ds[wayID][directID][attrID] = board_ds[wayID][directID][3 - attrID];
				}
			}
			__syncthreads();
			// calculate getscore
			if (directID<2){
				getScore_d(attr_d, attrPosition_ds, getScore_ds[wayID][directID], board_ds[wayID][directID], attrID);
				if ((wayID & 0x1) == 0 && (attrID & 0x3) == 0){
					getScore_ds[wayID][directID][attrID] += getScore_ds[wayID][directID][attrID + 1];
					getScore_ds[wayID][directID][attrID] += getScore_ds[wayID][directID][attrID + 2];
					getScore_ds[wayID][directID][attrID] += getScore_ds[wayID][directID][attrID + 3];
					getScore_ds[wayID][directID][attrID] += getScore_ds[wayID + 1][directID][attrID + 0];
					getScore_ds[wayID][directID][attrID] += getScore_ds[wayID + 1][directID][attrID + 1];
					getScore_ds[wayID][directID][attrID] += getScore_ds[wayID + 1][directID][attrID + 2];
					getScore_ds[wayID][directID][attrID] += getScore_ds[wayID + 1][directID][attrID + 3];
				}
			}
			__syncthreads();
			if (directID<2){
				if (wayID == 0 && attrID == 0){		// could be parallel for wayID
					float tmpScore = 0;
					for (int i = 0; i<8; i += 2){
						for (int j = 0; j<attrNum; j += 4){
							tmpScore += getScore_ds[i][directID][j];
						}
					}
					getScore_ds[0][directID][0] = tmpScore;
				}
			}
			__syncthreads();
			if (directID == 0){
				s[0] = getScore_ds[0][0][0];
				s[1] = getScore_ds[0][1][0];
				dif = s[1] + rec_d[activeID*recordSize + i].earned - s[0];
				updateAttr_d(attr_d, attrPosition_ds, board_ds[wayID][directID], attrID, dif*learnSpeed);
			}
			__syncthreads();
		}
	}
	if (threadIdx.x == 0){
		// restore the random seed
		int oldcnt = atomicAdd(&(learnCnt_d[2]), 1);
		randState[oldcnt % 1000] = randState_ds;
		// record the learning result
		oldcnt = atomicAdd(&(learnCnt_d[0]), 1);
		atomicAdd(&recProgress_d[oldcnt / 1000 * 3], score);
		atomicMax(&recProgress_d[oldcnt / 1000 * 3 + 1], score);
		atomicAdd(&recProgress_d[oldcnt / 1000 * 3 + 2], step);
	}
}

double run(vector<Attr> &attr, int times, float learnSpeed = 0, int *learnCnt_d = NULL, int *recProgress_d = NULL){
	/*
	bool learn = learnSpeed > 1e-6;
	MoveFunc moveArr[4];
	moveArr[0]=&board::up;
	moveArr[1]=&board::down;
	moveArr[2]=&board::left;
	moveArr[3]=&board::right;
	int maxscore=0, maxstep=0;
	int goal=0;
	*/
	double acc = 0;

	float *attr_d;
	int *attrPosition_d;
	hiprandState_t *randState;
	int *randSeed_d;
	record_d *rec_d;

	hipMalloc((void**)&attr_d, (1 << attrDataSize)*attr.size()*sizeof(float));
	hipMalloc((void**)&attrPosition_d, attr.size()*sizeof(int));
	hipMalloc((void**)&randState, branchSizeLimit*sizeof(hiprandState_t));
	hipMalloc((void**)&randSeed_d, branchSizeLimit*sizeof(int));
	hipMalloc((void**)&rec_d, branchSize*recordSize*sizeof(record_d));

	int randSeed_h[1000];
	srand(time(NULL));
	for (int i = 0; i<branchSizeLimit; i++){
		randSeed_h[i] = rand();
	}
	hipMemcpy(randSeed_d, randSeed_h, branchSizeLimit*sizeof(int), hipMemcpyHostToDevice);
	initrand << <branchSizeLimit, 1 >> >(randState, randSeed_d);

	for (int i = 0; i<attr.size(); i++){
		hipMemcpy(attr_d + i*(1 << attrDataSize), attr[i].data, (1 << attrDataSize)*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(attrPosition_d + i, &attr[i].position, sizeof(int), hipMemcpyHostToDevice);
	}
	// in compute capability 2.0, the grid X dimension has to be lower than 65535
	dim3 gridsize(10000, times / 10000);
	run_d << <gridsize, (8 * 4)*attr.size() >> >(attr_d, attrPosition_d, rec_d, randState, learnSpeed, learnCnt_d, recProgress_d);
	for (int i = 0; i<attr.size(); i++){
		hipMemcpy(attr[i].data, attr_d + i*(1 << attrDataSize), (1 << attrDataSize)*sizeof(float), hipMemcpyDeviceToHost);
	}

	hipFree(attr_d);
	hipFree(attrPosition_d);
	hipFree(randState);
	hipFree(randSeed_d);
	hipFree(rec_d);
	return acc / times;
}

int main(int argc, char* argv[]) {
	genMap();
	vector<Attr> attr;
	int learnTimes = 10000;
	double learnSpeed = 0.01;
	char in[2048], out[2048];
	if (argc >= 3) {
		strcpy(in, argv[1]);
		strcpy(out, argv[2]);
		if (argc >= 4)
			learnTimes = atoi(argv[3]);
		if (argc >= 5)
			learnSpeed = atof(argv[4]);
	}
	else {
		fprintf(stderr, "error: %s <input> <output> <learnTimes> <learnSpeed>\n", argv[0]);
		return 1;
	}
	srand(time(NULL));
	pair<clock_t, clock_t> real;
	real.first = clock();
	if (!load(in, attr)) {
		fprintf(stderr, "file open failed.\n");
		return 1;
	}
	hipDeviceProp_t DeviceProp;
	hipGetDeviceProperties(&DeviceProp, 0);
	double freq = DeviceProp.clockRate;
	freq *= 1000;
	int learnCnt_h[3] = { 0, 0, 0 };
	int *learnCnt_d;
	int *recProgress_d;
	for (int i = 0; i<10005; i++){
		recProgress_h[i][0] = 0,
			recProgress_h[i][1] = 0;
		recProgress_h[i][2] = 0;
	}
	hipMalloc((void**)&learnCnt_d, sizeof(int));
	hipMalloc((void**)&recProgress_d, 10005 * 3 * sizeof(int));
	hipMemcpy(learnCnt_d, &learnCnt_h, 3 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(recProgress_d, recProgress_h, 10005 * 3 * sizeof(int), hipMemcpyHostToDevice);
	pair<clock_t, clock_t> run_t;
	run_t.first = clock();
	run(attr, learnTimes, learnSpeed, learnCnt_d, recProgress_d);
	run_t.second = clock();
	hipMemcpy(recProgress_h, recProgress_d, 10005 * 3 * sizeof(int), hipMemcpyDeviceToHost);
	FILE *recfile = fopen("rec.csv", "w");
	long long sumRec2 = 0;
	for (int i = 0; i<learnTimes / 1000; i++){
		fprintf(recfile, "%10d,%10.3f,%10d,%10d\n", (i + 1) * 1000, (double)recProgress_h[i][0] / 1000, recProgress_h[i][1], recProgress_h[i][2]);
		sumRec2 += recProgress_h[i][2];
	}
	printf("Speed: %.2f steps/s\n", 1.*sumRec2 / (run_t.second - run_t.first) *CLOCKS_PER_SEC);
	printf("Training time: %.3f s\n", 1.*(run_t.second - run_t.first) / CLOCKS_PER_SEC);
	fclose(recfile);
	hipFree(learnCnt_d);
	hipFree(recProgress_d);
	if (!save(out, attr)) {
		fprintf(stderr, "file open failed.\n");
		return 1;
	}
	real.second = clock();
	printf("Real time: %.3f s\n", 1.*(real.second - real.first) / CLOCKS_PER_SEC);
	return 0;

}
